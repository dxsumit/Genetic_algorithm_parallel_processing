#include "hip/hip_runtime.h"
%%cu

// C++ program to create target string, starting from random string using Genetic Algorithm
 
#include "bits/stdc++.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
using namespace std;
 
// Number of individuals in each generation
#define POPULATION_SIZE 10
// #define POPULATION_SIZE 500000
int generation = 0;
 
// Valid Genes
#define GENES "`~1!2@3#4$5%6^7&8*9(0)-_+QWERTYUIOPASDFGHJKLZXCVBNMqwertyuiopasdfghjklzxcvbnm[{]}|;:'\",./?< >"
#define genesSize 93

// Target string to be generated
#define TARGET "Random Generation..."
#define targetSize 20


// // Create random genes for mutation
// char mutated_genes() {
//     int len = GENES.size();
//     return GENES[rand()%len];
// }
 
// // create chromosome or string of genes
// string create_gnome() {
//     int len = TARGET.size();
//     string gnome = "";
//     for(int i=0; i<len; i++)
//         gnome += mutated_genes();
//     return gnome;
// }
 
// Class representing individual in population
class Individual {

    public:
        int fitness;
        char chromosome[targetSize];
        
        __device__ __host__ Individual(char *chromosome);  
};
 
Individual::Individual(char *chromosome) {
    // this->chromosome = chromosome;
    for (int i=0; chromosome[i] != '\0'; i++) {
        this->chromosome[i] = chromosome[i];
    }
   
    // int len = TARGET.size();
    int offspring_fitness = 0;
    for(int i=0; i<targetSize; i++) {
        if(chromosome[i] != TARGET[i]){
            offspring_fitness++;
        }
    }
    this->fitness = offspring_fitness;  
};
 
// Overloading < operator
bool compare(Individual* ind1, Individual* ind2) {
    return ind1->fitness < ind2->fitness;
}
 

__device__ float generate(hiprandState* globalState, int ind) {
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed ) {
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}


// hiprandState* globalState for generating random...
__global__ void population_kernel(Individual **population, hiprandState *globalState) {

    // printf("Hello..");
    
    int index = threadIdx.x + blockIdx.x * 5;       // M is number of blocks..

    int number;
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // create initial population
   
    // for(int i=0; i<POPULATION_SIZE; i++) {
    //     char gnome[targetSize];
    //     for(int j=0; j<targetSize; j++){
    //         number = generate(globalState, id)*genesSize;
    //         gnome[j] = GENES[number];
    //     }
  
    //     population[i] = new Individual(gnome);
 
    //     // for(int j=0; j<targetSize; j++){
    //     //     printf("%c", population[i]->chromosome[j]);
    //     // }
    //     // printf(" --- Fitness %d\n", population[i]->fitness);
    // }
    // // printf("\nComplete., %d", population[0]->fitness);


    if(index < POPULATION_SIZE){

        char gnome[targetSize];
        for(int j=0; j<targetSize; j++){
            number = generate(globalState, id)*genesSize;
            gnome[j] = GENES[number];
        }
  
        population[index] = new Individual(gnome);
 
        for(int j=0; j<targetSize; j++){
            printf("%c", population[index]->chromosome[j]);
        }
        printf(" --- Fitness %d\n", population[index]->fitness);

    }

    // printf("Completed...\n");



}



// Driver code
int main() {

    // cout << "\nfine";

    int size = POPULATION_SIZE * sizeof(Individual);       /////////////
    Individual **population;           // host copies        ////
    Individual **d_population;          // device copies
    
    // allocate space for host...
    population = (Individual**)malloc(size);                /////

    // allocate space for device copies
    hipMalloc((void **)&d_population, size);


    hiprandState* devStates;
    hipMalloc (&devStates, POPULATION_SIZE * sizeof(hiprandState));
    srand(time(0));
    /** ADD THESE TWO LINES **/
    int seed = rand();
    setup_kernel<<<2, 5>>>(devStates,seed);
    /** END ADDITION **/


    // copy data to device..
    hipMemcpy(d_population, population, size, hipMemcpyHostToDevice);

    // Launch kernel on GPU with N blocks
    population_kernel<<<5,2>>>(d_population, devStates);
    hipDeviceSynchronize();

    // copy result back to host..
    hipMemcpy(population, d_population, size, hipMemcpyDeviceToHost);

    cout << "\nfine..\n";
    cout << "Value " << population[0]->fitness;

    // for(int index=0; index<POPULATION_SIZE; index++){

    //     cout << "Inside..\n";
    //     for(int j=0; j<targetSize; j++){
    //         cout << population[index]->chromosome[j];
    //     }
    //     cout << " --- Fitness " << population[index]->fitness << endl;

    // }

    


    // int threads;
    // bool found = false;
 
    // // create initial population
    // for(int i=0; i<POPULATION_SIZE; i++) {
    //     // string gnome = create_gnome();

    //     int len = targetSize;
    //     string gnome = "";
    //     for(int i=0; i<len; i++)
    //         gnome += GENES[rand()%genesSize];
            
        
    //     // population[i] = new Individual(gnome);
    // }
    

    // while(!found) {

    //     // sort the population in increasing order of fitness score
    //     sort(population.begin(), population.end(), compare);

    //     // if the individual having lowest fitness score ie. 0 then we know that we have reached to the target and break the loop
    //     if(population[0]->fitness <= 0){
    //         found = true;
    //         break;
    //     }

    //     // Otherwise generate new offsprings for new generation
    //     vector<Individual*> new_generation(POPULATION_SIZE);
    
    //     // Perform Elitism, that mean 10% of fittest population goes to the next generation
    //     int s = (10*POPULATION_SIZE)/100;

    //     for(int i=0; i<s; i++){
    //         new_generation[i] = population[i];
    //     }

    //     // From 50% of fittest population, Individuals will mate to produce offspring
    //     int right = (50*POPULATION_SIZE)/100;
        
    //     for(int i=s; i<POPULATION_SIZE; i++) {

    //         // int r = random_num(0, right);
    //         int r = rand()%(right+1);
    //         Individual* parent1 = population[r];

    //         // r = random_num(0, right);
    //         r = rand()%(right+1);
    //         Individual* parent2 = population[r];

    //         // chromosome for offspring
    //         string child_chromosome = "";
    //         string chromosome = parent1->chromosome;
    //         int len = chromosome.size();

    //         for(int i = 0;i<len;i++) {
    //             // random probability
    //             float p = (rand()%101)/100;
        
    //             // if prob is less than 0.45, insert gene from parent 1
    //             if(p < 0.45)
    //                 child_chromosome += chromosome[i];
    //             // if prob is between 0.45 and 0.90, insertgene from parent 2
    //             else if(p < 0.90)
    //                 child_chromosome += parent2->chromosome[i];
    //             // otherwise insert random gene(mutate), for maintaining diversity
    //             else
    //                 child_chromosome += mutated_genes();
    //         }
        
    //         // create new Individual(offspring) using generated chromosome for offspring
    //         Individual* offspring = new Individual(child_chromosome);

    //         new_generation[i] = offspring;
    //     }

    //     population = new_generation;
    //     cout<< "Generation: " << generation << "\t";
    //     cout<< "String: "<< population[0]->chromosome <<"\t";
    //     cout<< "Fitness: "<< population[0]->fitness << "\n";

    //     generation++;

    // }

    // cout<< "Generation: " << generation << "\t";
    // cout<< "String: "<< population[0]->chromosome <<"\t";
    // cout<< "Fitness: "<< population[0]->fitness << "\n";

   return 0;

}